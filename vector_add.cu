
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    result[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);


  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<N,1>>>(c, a, b, N);
  hipDeviceSynchronize();
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
