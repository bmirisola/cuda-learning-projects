
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  /*
   * Assign values to these variables so that the output string below prints the
   * requested properties of the currently active GPU.
   */

  /*
   * Device ID is required first to query the device.
   */

  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  /*
   * `props` now contains several properties about the current device.
   */
  
  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

  /*
   * There should be no need to modify the output string below.
   */

  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}