
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void someKernel(){
	printf("gridDim.x : %d\n",gridDim.x);
	printf("gridDim.y : %d\n", gridDim.y);
	printf("blockDim.x: %d\n", blockDim.x);
	printf("blockDim.y: %d\n",blockDim.y);
	//printf(": %d",);
	//printf(": %d",);

}

int main(){
	
	dim3 threads_per_block(1, 1, 1);
	dim3 number_of_blocks(16, 16, 1);
	someKernel<<<number_of_blocks, threads_per_block>>>();
	hipDeviceSynchronize();
}
