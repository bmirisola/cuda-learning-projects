
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void forloop(){
    printf("%i\n",threadIdx.x) ;
}

int main() {
    //calls for loop with 1000 blocks and 500 threads
    forloop<<<1000,500>>>();
    hipDeviceSynchronize();
    return 0;
}
